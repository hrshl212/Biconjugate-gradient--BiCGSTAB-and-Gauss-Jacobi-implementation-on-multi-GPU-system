#include "hip/hip_runtime.h"
// multi_gpu_bicg_time.cu - Solves transient heat equation using true BiCGSTAB solver (Krylov) for implicit scheme on multi-GPU (MPI + CUDA)

#include <cstdio>
#include <cmath>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <chrono>

#define IDX(i, j, nx) ((i) + (j)*(nx))

__global__
void apply_operator(const float* u, float* Au, int nx, int ny, float dx2, float dy2, float alpha_dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i < nx-1 && j < ny-1) {
        int id = IDX(i, j, nx);
        float uij = u[id];
        float lap = (u[IDX(i-1,j,nx)] + u[IDX(i+1,j,nx)] - 2*uij) / dx2
                  + (u[IDX(i,j-1,nx)] + u[IDX(i,j+1,nx)] - 2*uij) / dy2;
        Au[id] = uij - alpha_dt * lap;
    }
}

void exchange_halo(float* d_field, int nx, int ny_local, int rank, int size) {
    MPI_Request reqs[4];
    int tag0 = 0, tag1 = 1;

    float* send_top = d_field + IDX(0, ny_local - 2, nx);
    float* recv_top = d_field + IDX(0, ny_local - 1, nx);
    float* send_bot = d_field + IDX(0, 1, nx);
    float* recv_bot = d_field + IDX(0, 0, nx);

    int n_req = 0;

    if (rank > 0) {
        MPI_Irecv(recv_bot, nx, MPI_FLOAT, rank - 1, tag0, MPI_COMM_WORLD, &reqs[n_req++]);
        MPI_Isend(send_bot, nx, MPI_FLOAT, rank - 1, tag1, MPI_COMM_WORLD, &reqs[n_req++]);
    }

    if (rank < size - 1) {
        MPI_Irecv(recv_top, nx, MPI_FLOAT, rank + 1, tag1, MPI_COMM_WORLD, &reqs[n_req++]);
        MPI_Isend(send_top, nx, MPI_FLOAT, rank + 1, tag0, MPI_COMM_WORLD, &reqs[n_req++]);
    }

    MPI_Waitall(n_req, reqs, MPI_STATUSES_IGNORE);
}

__global__
void vec_axpby(float* z , const float* x, const float* y, float alpha, float beta, int nx, int ny_local) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = nx * ny_local;
    if (idx < N) {
        int j = idx / nx;
        if (j > 0 && j < ny_local - 1)  // skip halos
            z[idx] = alpha * x[idx] + beta * y[idx];
    }
}

__global__
void vec_add(float* z, const float* x, const float* y, float alpha, float beta, int nx, int ny_local) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = nx * ny_local;
    if (idx < N) {
        int j = idx / nx;
        if (j > 0 && j < ny_local - 1)  // skip halos
            z[idx] =  alpha * x[idx] + beta * y[idx];
    }
}

__global__
void dot_product_kernel(const float* x, const float* y, const int* mask, float* partial_sum, int nx, int ny_local, int rank, int size) {
    extern __shared__ float cache[];  // dynamic shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float temp = 0;
    int total = nx * ny_local;

    if (i < total) temp = mask[i] * x[i] * y[i];

    cache[tid] = temp;
    __syncthreads();
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) cache[tid] += cache[tid + stride];
        __syncthreads();
    }
    if (tid == 0) partial_sum[blockIdx.x] = cache[0];

}

float dot_product(const float* d_x, const float* d_y, const int* d_mask, float* d_partial, float* h_partial, int nx, int ny_local, int rank, int size) {
    int threads = std::min(256, nx* ny_local);
    int blocks = (nx* ny_local + threads - 1) / threads;
    
    int shared_mem_size = threads * sizeof(float);

    dot_product_kernel<<<blocks, threads, shared_mem_size>>>(d_x, d_y, d_mask, d_partial, nx, ny_local, rank, size);
    hipMemcpy(h_partial, d_partial, blocks * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < blocks; ++i) sum += h_partial[i];
    float global_sum;
    MPI_Allreduce(&sum, &global_sum, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
    return global_sum;
}

__global__
void enforce_neumann_bc_x(float* u, int nx, int ny_local, int rank, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nx) {
        if (rank == 0) u[IDX(i, 0, nx)] = u[IDX(i, 1, nx)];
        if (rank == size - 1) u[IDX(i, ny_local - 1, nx)] = u[IDX(i, ny_local - 2, nx)];
    }
}

__global__
void enforce_neumann_bc_y(float* u, int nx, int ny_local) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < ny_local) {
        u[IDX(0, j, nx)] = u[IDX(1, j, nx)];
        u[IDX(nx - 1, j, nx)] = u[IDX(nx - 2, j, nx)];
    }
}

void enforce_neumann_bc(float* d_u, int nx, int ny_local, int rank, int size) {
    int threads = 256;
    int blocks_x = (nx + threads - 1) / threads;
    int blocks_y = (ny_local + threads - 1) / threads;
    enforce_neumann_bc_x<<<blocks_x, threads>>>(d_u, nx, ny_local, rank, size);
    enforce_neumann_bc_y<<<blocks_y, threads>>>(d_u, nx, ny_local);
}

void initialize_gaussian(float* u, int nx, int ny_local, int global_j_start, int ny_total) {
    float cx = nx / 2;
    float cy = ny_total / 2;
    float sigma = 10.0f;
    for (int j = 0; j < ny_local; ++j) {
        for (int i = 0; i < nx; ++i) {
            float dx = i - cx;
            float dy = (j + global_j_start - 1) - cy;
            u[IDX(i, j, nx)] = expf(-(dx*dx + dy*dy)/(2*sigma*sigma));
        }
    }
}

void save_solution(float* d_u, int nx, int ny_local, int rank, int size, int step, int ny_total) {
    float* h_local = new float[nx * (ny_local - 2)];
    hipMemcpy(h_local, d_u + nx, nx * (ny_local - 2) * sizeof(float), hipMemcpyDeviceToHost);
    float* h_global = nullptr;
    if (rank == 0) h_global = new float[nx * ny_total];

    MPI_Gather(h_local, nx * (ny_local - 2), MPI_FLOAT,
               h_global, nx * (ny_total / size), MPI_FLOAT,
               0, MPI_COMM_WORLD);

    if (rank == 0) {
        char fname[64];
        sprintf(fname, "heat_%04d.npy", step);
        FILE* f = fopen(fname, "wb");
        fwrite(h_global, sizeof(float), nx * ny_total, f);
        fclose(f);
        printf("Saved %s\n", fname);
        delete[] h_global;
    }
    delete[] h_local;
}

void print_mat(float* d_u, int nx, int ny_local, int rank, int size,  int ny_total) {
    float* h_local = new float[nx * (ny_local - 2)];
    hipMemcpy(h_local, d_u + nx, nx * (ny_local - 2) * sizeof(float), hipMemcpyDeviceToHost);
    float* h_global = nullptr;
    if (rank == 0) h_global = new float[nx * ny_total];

    MPI_Gather(h_local, nx * (ny_local - 2), MPI_FLOAT,
               h_global, nx * (ny_total / size), MPI_FLOAT,
               0, MPI_COMM_WORLD);
    
    std::cout << "printing mat" << std::endl;
    if(rank == 0){
        for(int i=0; i< nx; i++){
            for(int j=0; j<ny_total; j++){
                std::cout << h_global[IDX(i,j,nx)] << "   ";
            }
            std::cout << std::endl;
        }
    }
    delete[] h_global;
    delete[] h_local;
}

void create_mask(int* mask, int nx, int ny_local, int rank, int size) {
    for (int j = 0; j < ny_local; ++j) {
        for (int i = 0; i < nx; ++i) {
            bool is_boundary = (i == 0 || i == nx - 1 ||
                               (j == 0 && rank == 0) ||
                               (j == ny_local - 1 && rank == size - 1));
            mask[IDX(i, j, nx)] = is_boundary ? 0 : 1;
        }
    }
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    hipSetDevice(rank);
    const int nx = 128, ny_total = 128;
    const float alpha_e = 0.1f, dx = 1.0f, dy = 1.0f;
    const float dt = 1.0f, alpha_dt = alpha_e * dt;
    const int steps = 1000, max_iters = 500, save_interval = 40;
    const float tol = 1e-6f;

    int ny_local = ny_total / size + 2;
    int N = nx * ny_local;
    size_t bytes = N * sizeof(float);
    int* h_mask = new int[nx * ny_local];
    create_mask(h_mask, nx, ny_local, rank, size);

    float *d_x, *d_rhs, *d_r, *d_r0, *d_p, *d_v, *d_Ap, *d_s, *d_tt;
    float *d_partial;
    int* d_mask;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_rhs, bytes);
    hipMalloc(&d_r, bytes);
    hipMalloc(&d_r0, bytes);
    hipMalloc(&d_p, bytes);
    hipMalloc(&d_v, bytes);
    hipMalloc(&d_Ap, bytes);
    hipMalloc(&d_s, bytes);
    hipMalloc(&d_tt, bytes);
    hipMalloc(&d_mask, nx * ny_local * sizeof(int));
    hipMalloc(&d_partial, ((N + 255) / 256) * sizeof(float));
    float* h_partial = new float[(N + 255) / 256];
    hipMemcpy(d_mask, h_mask, nx * ny_local * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_mask;

    float* h_u_init = new float[N];
    initialize_gaussian(h_u_init, nx, ny_local, rank * (ny_total / size), ny_total);
    hipMemcpy(d_x, h_u_init, bytes, hipMemcpyHostToDevice);
    delete[] h_u_init;

    std::cout << std::fixed << std::setprecision(1); // Set output to fixed-point notation with 1 decimal places
    // print_mat(d_x,  nx,  ny_local,  rank,  size,  ny_total);

    int threads_one_dir = 16;
    dim3 threads(threads_one_dir, threads_one_dir);
    dim3 blocks((nx - 2 + threads_one_dir-1) / threads_one_dir, (ny_local - 2 + threads_one_dir-1) / threads_one_dir);
    dim3 threads1D(threads_one_dir*threads_one_dir);
    dim3 blocks1D((N + (threads_one_dir*threads_one_dir-1)) / (threads_one_dir*threads_one_dir));

    auto start = std::chrono::high_resolution_clock::now();

    for (int t = 0; t <= steps; ++t) {
        hipMemcpy(d_rhs, d_x, bytes, hipMemcpyDeviceToDevice);

        // hipMemset(d_x, 0, bytes);  // or d_x = d_rhs if warm-starting
        // r = b - A*u
        exchange_halo(d_x, nx, ny_local, rank, size);    
        enforce_neumann_bc(d_x, nx, ny_local, rank, size);
        apply_operator<<<blocks, threads>>>(d_x, d_Ap, nx, ny_local, dx*dx, dy*dy, alpha_dt);

        vec_add<<<blocks1D, threads1D>>>(d_r, d_rhs, d_Ap, 1.0f, -1.0f,  nx, ny_local);

        hipMemcpy(d_r0, d_r, bytes, hipMemcpyDeviceToDevice); //d_r0 is the shadow residual
        hipMemcpy(d_p, d_r, bytes, hipMemcpyDeviceToDevice);
        hipMemcpy(d_v, d_r, bytes, hipMemcpyDeviceToDevice); 

        float rho_old = 1.0f, alpha = 1.0f, omega = 1.0f;
        float beta, rho_new, resid, denom;
        
        if (t % save_interval == 0)
            save_solution(d_x, nx, ny_local, rank, size, t, ny_total);

        int k = 0;
        while (k < max_iters) {
            rho_new = dot_product(d_r0, d_r, d_mask, d_partial, h_partial, nx, ny_local, rank, size);
            beta = (rho_new / rho_old) * (alpha / omega);

            // p = r + beta * (p - omega * v)
            vec_axpby<<<blocks1D, threads1D>>>(d_p, d_p, d_v, 1.0f, -omega, nx, ny_local);
            vec_axpby<<<blocks1D, threads1D>>>(d_p, d_p, d_r, beta, 1.0f,  nx, ny_local);


            // v = A * p
            exchange_halo(d_p, nx, ny_local, rank, size);    
            enforce_neumann_bc(d_p, nx, ny_local, rank, size);
            apply_operator<<<blocks, threads>>>(d_p, d_v, nx, ny_local, dx*dx, dy*dy, alpha_dt);

            denom = dot_product(d_r0, d_v, d_mask, d_partial, h_partial, nx, ny_local, rank, size);
            alpha = rho_new / denom;

            // s = r - alpha * v
            vec_axpby<<<blocks1D, threads1D>>>(d_s, d_r, d_v, 1.0f, -alpha,  nx, ny_local);   

            // t = A * s
            exchange_halo(d_s, nx, ny_local, rank, size);    
            enforce_neumann_bc(d_s, nx, ny_local, rank, size);
            apply_operator<<<blocks, threads>>>(d_s, d_tt, nx, ny_local, dx*dx, dy*dy, alpha_dt); 

            omega = dot_product(d_tt, d_s, d_mask, d_partial, h_partial, nx, ny_local, rank, size) / dot_product(d_tt, d_tt, d_mask, d_partial, h_partial, nx, ny_local, rank, size);

            // x = x + alpha * p + omega * s
            vec_axpby<<<blocks1D, threads1D>>>(d_p, d_p, d_s, alpha, omega, nx, ny_local);
            vec_axpby<<<blocks1D, threads1D>>>(d_x, d_x, d_p, 1.0f, 1.0f, nx, ny_local);
    
            // r = s - omega * t
            vec_axpby<<<blocks1D, threads1D>>>(d_r, d_s, d_tt, 1.0f, -omega, nx, ny_local);
    
            resid = sqrt(dot_product(d_r, d_r, d_mask, d_partial, h_partial, nx, ny_local, rank, size));
            // printf("Iteration %d, Residual = %e\n", k, resid);
            if (resid < tol) break;
    
            rho_old = rho_new;
            k++;
        }

        // if (rank == 0 && t % save_interval == 0)
        //     printf("[t=%d] BiCG converged in %d iterations\n", t, k);
    }

    // Record the end time
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate the duration
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Print the execution time
    std::cout << "Execution time: " << duration.count() << " microseconds" << std::endl;

    hipFree(d_x); hipFree(d_rhs); hipFree(d_r); hipFree(d_r0);
    hipFree(d_p); hipFree(d_v); hipFree(d_Ap); hipFree(d_s); hipFree(d_partial);
    hipFree(d_mask); hipFree(d_tt);
    delete[] h_partial;
    MPI_Finalize();
    return 0;
}

